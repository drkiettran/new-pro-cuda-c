#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <fstream>

const int MAXDATASIZE = 1024 * 1024;

// Kernel for accessing mapped memory
__global__ void accessMappedMemoryKernel(int* mappedData, int dataSize) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < dataSize) {
        mappedData[idx] += 1;
    }
}

void testMemoryCopySpeed(int iter, int step, const char* filename) {
    hipStream_t str;
    int* h_data_pageable, * h_data_pinned, * h_data_mapped;
    int* d_data;
    int i, dataSize;
    hipEvent_t startT, endT;
    float duration;

    // Allocate pageable memory
    h_data_pageable = (int*)malloc(sizeof(int) * MAXDATASIZE);

    // Allocate pinned memory
    hipHostMalloc((void**)&h_data_pinned, sizeof(int) * MAXDATASIZE);

    // Allocate mapped memory
    hipHostAlloc((void**)&h_data_mapped, sizeof(int) * MAXDATASIZE, hipHostMallocMapped);

    // Allocate device memory
    hipMalloc((void**)&d_data, sizeof(int) * MAXDATASIZE);

    // Initialize host data
    for (i = 0; i < MAXDATASIZE; i++) {
        h_data_pageable[i] = h_data_pinned[i] = h_data_mapped[i] = i;
    }

    // Create CUDA events and stream
    hipEventCreate(&startT);
    hipEventCreate(&endT);
    hipStreamCreate(&str);

    std::ofstream outFile(filename);
    outFile << "DataSize(Bytes),Pageable_H2D(MB/s),Pageable_D2H(MB/s),Pinned_H2D(MB/s),Pinned_D2H(MB/s),Mapped_H2D(MB/s),Mapped_D2H(MB/s)\n";

    for (dataSize = step; dataSize <= MAXDATASIZE; dataSize += step) {
        float pageableH2D, pageableD2H;
        float pinnedH2D, pinnedD2H;
        float mappedH2D, mappedD2H;

        // Test pageable H2D
        hipEventRecord(startT, str);
        for (i = 0; i < iter; i++) {
            hipMemcpyAsync(d_data, h_data_pageable, sizeof(int) * dataSize, hipMemcpyHostToDevice, str);
        }
        hipEventRecord(endT, str);
        hipEventSynchronize(endT);
        hipEventElapsedTime(&duration, startT, endT);
        pageableH2D = (dataSize * sizeof(int) * iter / (duration / 1e3)) / (1024 * 1024);

        // Test pageable D2H
        hipEventRecord(startT, str);
        for (i = 0; i < iter; i++) {
            hipMemcpyAsync(h_data_pageable, d_data, sizeof(int) * dataSize, hipMemcpyDeviceToHost, str);
        }
        hipEventRecord(endT, str);
        hipEventSynchronize(endT);
        hipEventElapsedTime(&duration, startT, endT);
        pageableD2H = (dataSize * sizeof(int) * iter / (duration / 1e3)) / (1024 * 1024);

        // Test pinned H2D
        hipEventRecord(startT, str);
        for (i = 0; i < iter; i++) {
            hipMemcpyAsync(d_data, h_data_pinned, sizeof(int) * dataSize, hipMemcpyHostToDevice, str);
        }
        hipEventRecord(endT, str);
        hipEventSynchronize(endT);
        hipEventElapsedTime(&duration, startT, endT);
        pinnedH2D = (dataSize * sizeof(int) * iter / (duration / 1e3)) / (1024 * 1024);

        // Test pinned D2H
        hipEventRecord(startT, str);
        for (i = 0; i < iter; i++) {
            hipMemcpyAsync(h_data_pinned, d_data, sizeof(int) * dataSize, hipMemcpyDeviceToHost, str);
        }
        hipEventRecord(endT, str);
        hipEventSynchronize(endT);
        hipEventElapsedTime(&duration, startT, endT);
        pinnedD2H = (dataSize * sizeof(int) * iter / (duration / 1e3)) / (1024 * 1024);

        // Test mapped H2D (access by kernel)
        int* d_mapped;
        hipHostGetDevicePointer((void**)&d_mapped, h_data_mapped, 0);
        hipEventRecord(startT, str);
        for (i = 0; i < iter; i++) {
            accessMappedMemoryKernel << <(dataSize + 255) / 256, 256, 0, str >> > (d_mapped, dataSize);
        }
        hipEventRecord(endT, str);
        hipEventSynchronize(endT);
        hipEventElapsedTime(&duration, startT, endT);
        mappedH2D = (dataSize * sizeof(int) * iter / (duration / 1e3)) / (1024 * 1024);

        // Test mapped D2H (access by kernel)
        hipEventRecord(startT, str);
        for (i = 0; i < iter; i++) {
            accessMappedMemoryKernel << <(dataSize + 255) / 256, 256, 0, str >> > (d_mapped, dataSize);
        }
        hipEventRecord(endT, str);
        hipEventSynchronize(endT);
        hipEventElapsedTime(&duration, startT, endT);
        mappedD2H = (dataSize * sizeof(int) * iter / (duration / 1e3)) / (1024 * 1024);

        // Write results to file
        outFile << (dataSize * sizeof(int)) << "," << pageableH2D << "," << pageableD2H << "," << pinnedH2D << "," << pinnedD2H << "," << mappedH2D << "," << mappedD2H << "\n";
    }

    outFile.close();

    // Clean up
    hipStreamDestroy(str);
    hipEventDestroy(startT);
    hipEventDestroy(endT);
    free(h_data_pageable);
    hipHostFree(h_data_pinned);
    hipHostFree(h_data_mapped);
    hipFree(d_data);
    hipDeviceReset();
}

int main(int argc, char** argv) {
    if (argc < 3) {
        printf("Usage: %s <iterations> <step size>\n", argv[0]);
        return -1;
    }

    int iter = atoi(argv[1]);
    int step = atoi(argv[2]);

    testMemoryCopySpeed(iter, step, "memory_copy_results.csv");

    printf("Memory copy speed test completed. Results saved to memory_copy_results.csv.\n");
    return 0;
}
