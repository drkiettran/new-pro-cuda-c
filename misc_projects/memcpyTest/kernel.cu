﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

// example of command line arguments: 4 128
const int MAXDATASIZE = 1024 * 1024;
int main(int argc, char** argv) {
	int iter = atoi(argv[1]);
	int step = atoi(argv[2]);
	hipStream_t str;
	int* h_data, * d_data;
	int i, dataSize;
	hipEvent_t startT, endT;
	float duration;

	hipHostMalloc((void**)&h_data, sizeof(int) * MAXDATASIZE, hipHostMallocDefault);
	hipMalloc((void**)&d_data, sizeof(int) * MAXDATASIZE);
	for (i = 0; i < MAXDATASIZE; i++) {
		h_data[i] = i;
	}

	hipEventCreate(&startT);
	hipEventCreate(&endT);
	hipStreamCreate(&str);
	for (dataSize = 0; dataSize <= MAXDATASIZE; dataSize += step) {
		hipEventRecord(startT, str);
		for (i = 0; i < iter; i++) {
			hipMemcpyAsync(d_data, h_data, sizeof(int) * dataSize, hipMemcpyHostToDevice, str);
		}
		hipEventRecord(endT, str);
		hipEventSynchronize(endT);
		hipEventElapsedTime(&duration, startT, endT);
		printf("%i %f\n", (int)(dataSize * sizeof(int)), duration / iter);

	}
	hipStreamDestroy(str);
	hipEventDestroy(startT);
	hipEventDestroy(endT);
	hipHostFree(h_data);
	hipFree(d_data);
	hipDeviceReset();
	return 0;
}