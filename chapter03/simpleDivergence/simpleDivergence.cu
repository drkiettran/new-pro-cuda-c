#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"


/*
 * simpleDivergence demonstrates divergent code on the GPU and its impact on
 * performance and CUDA metrics.
 */

__global__ void mathKernel1(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if (tid % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel2(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    //if ((tid / warpSize) % 2 == 0)
    // if ((tid >> 5) % 2 == 0)
    if (((tid >> 5) & 0x01) == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel3(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    bool ipred = (tid % 2 == 0);

    if (ipred)
    {
        ia = 100.0f;
    }

    if (!ipred)
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void mathKernel4(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    int itid = tid >> 5;

    if (itid & 0x01 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}

__global__ void warmingup(float* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;

    if ((tid / warpSize) % 2 == 0)
    {
        ia = 100.0f;
    }
    else
    {
        ib = 200.0f;
    }

    c[tid] = ia + ib;
}


int main(int argc, char** argv)
{
    std::chrono::steady_clock::time_point begin;

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s using Device %d: %s\n", argv[0], dev, deviceProp.name);

    // set up data size
    int size = 64;
    int blocksize = 64;

    if (argc > 1) blocksize = atoi(argv[1]);

    if (argc > 2) size = atoi(argv[2]);

    printf("Data size %d ", size);

    // set up execution configuration
    dim3 block(blocksize, 1);
    dim3 grid((size + block.x - 1) / block.x, 1);
    printf("Execution Configure (block %d grid %d)\n", block.x, grid.x);

    // allocate gpu memory
    float* d_C;
    size_t nBytes = size * sizeof(float);
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // run a warmup kernel to remove overhead
    CHECK(hipDeviceSynchronize());

    begin = StartTimer();
    warmingup << <grid, block >> > (d_C);
    CHECK(hipDeviceSynchronize());
    std::cout << "- grid.x: " << grid.x << std::endl << "- block.x: " << block.x << std::endl;
    std::cout << "Warming up on GPU: " << GetDurationInMicroSeconds(begin, StopTimer()) << " mcs" << std::endl;

    CHECK(hipGetLastError());

    // run kernel 1
    begin = StartTimer();
    mathKernel1 << <grid, block >> > (d_C);
    std::cout << "Running mathKernel1 on GPU: " << GetDurationInMicroSeconds(begin, StopTimer()) << " mcs" << std::endl;

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // run kernel 2
    begin = StartTimer();
    mathKernel2 << <grid, block >> > (d_C);
    std::cout << "Running mathKernel2 on GPU: " << GetDurationInMicroSeconds(begin, StopTimer()) << " mcs" << std::endl;
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // run kernel 3
    begin = StartTimer();
    mathKernel3 << <grid, block >> > (d_C);
    std::cout << "Running mathKernel3 on GPU: " << GetDurationInMicroSeconds(begin, StopTimer()) << " mcs" << std::endl;
  
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // run kernel 4
    begin = StartTimer();
    mathKernel4 << <grid, block >> > (d_C);
    std::cout << "Running mathKernel4 on GPU: " << GetDurationInMicroSeconds(begin, StopTimer()) << " mcs" << std::endl;

    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    // free gpu memory and reset divece
    CHECK(hipFree(d_C));
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}