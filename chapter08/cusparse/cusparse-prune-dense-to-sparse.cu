﻿/*
* From Chapter 16 - Appendix B: Examples of prune
* Example 16.1 - Prune dense to sparse
* 
* How to compile (assume cuda is installed at /usr/local/cuda/)
* nvcc -c -I/usr/local/cuda/include prunedense_example.cpp
* g++ -o prunedense_example.cpp prunedense_example.o -L/usr/local/cuda/lib64 -lcusparse -lcudart
*/
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

void printMatrix(int m, int n, const float* A, int lda, const char* name)
{
	for (int row = 0; row < m; row++) {
		for (int col = 0; col < n; col++) {
			float Areg = A[row + col * lda];
			printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
		}
	}
}
void printCsr(
	int m,
	int n,
	int nnz,
	const hipsparseMatDescr_t descrA,
	const float* csrValA,
	const int* csrRowPtrA,
	const int* csrColIndA,
	const char* name)
{
	const int base = (hipsparseGetMatIndexBase(descrA) != HIPSPARSE_INDEX_BASE_ONE) ?
		0 : 1;
	printf("matrix %s is %d-by-%d, nnz=%d, base=%d\n", name, m, n, nnz, base);
	for (int row = 0; row < m; row++) {
		const int start = csrRowPtrA[row] - base;
		const int end = csrRowPtrA[row + 1] - base;
		for (int colidx = start; colidx < end; colidx++) {
			const int col = csrColIndA[colidx] - base;
			const float Areg = csrValA[colidx];
			printf("%s(%d,%d) = %f\n", name, row + 1, col + 1, Areg);
		}
	}
}
int main(int argc, char* argv[])
{
	hipsparseHandle_t handle = NULL;
	hipStream_t stream = NULL;
	hipsparseMatDescr_t descrC = NULL;
	hipsparseStatus_t status = HIPSPARSE_STATUS_SUCCESS;
	hipError_t cudaStat1 = hipSuccess;
	hipError_t cudaStat2 = hipSuccess;
	hipError_t cudaStat3 = hipSuccess;
	hipError_t cudaStat4 = hipSuccess;
	hipError_t cudaStat5 = hipSuccess;
	const int m = 4;
	const int n = 4;
	const int lda = m;
	/*
	* | 1 0 2 -3 |
	* | 0 4 0 0 |
	* A = | 5 0 6 7 |
	* | 0 8 0 9 |
	*
	*/
	const float A[lda * n] = { 1, 0, 5, 0, 0, 4, 0, 8, 2, 0, 6, 0, -3, 0, 7, 9 };
	int* csrRowPtrC = NULL;
	int* csrColIndC = NULL;
	float* csrValC = NULL;
	float* d_A = NULL;
	int* d_csrRowPtrC = NULL;
	int* d_csrColIndC = NULL;
	float* d_csrValC = NULL;
	size_t lworkInBytes = 0;
	char* d_work = NULL;
	int nnzC = 0;
	float threshold = 4.1; /* remove Aij <= 4.1 */
	// float threshold = 0; /* remove zeros */
	printf("example of pruneDense2csr \n");
	printf("prune |A(i,j)| <= threshold \n");
	printf("threshold = %E \n", threshold);
	printMatrix(m, n, A, lda, "A");
	/* step 1: create cusparse handle, bind a stream */
	cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
	assert(hipSuccess == cudaStat1);
	status = hipsparseCreate(&handle);
	assert(HIPSPARSE_STATUS_SUCCESS == status);
	status = hipsparseSetStream(handle, stream);
	assert(HIPSPARSE_STATUS_SUCCESS == status);
	/* step 2: configuration of matrix C */
	status = hipsparseCreateMatDescr(&descrC);
	assert(HIPSPARSE_STATUS_SUCCESS == status);
	hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
	cudaStat1 = hipMalloc((void**)&d_A, sizeof(float) * lda * n);
	cudaStat2 = hipMalloc((void**)&d_csrRowPtrC, sizeof(int) * (m + 1));
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	/* step 3: query workspace */
	cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * lda * n, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);
	status = hipsparseSpruneDense2csr_bufferSizeExt(
		handle,
		m,
		n,
		d_A,
		lda,
		&threshold,
		descrC,
		d_csrValC,
		d_csrRowPtrC,
		d_csrColIndC,
		&lworkInBytes);
	assert(HIPSPARSE_STATUS_SUCCESS == status);
	printf("lworkInBytes (prune) = %lld \n", (long long)lworkInBytes);
	if (NULL != d_work) { hipFree(d_work); }
	cudaStat1 = hipMalloc((void**)&d_work, lworkInBytes);
	assert(hipSuccess == cudaStat1);
	/* step 4: compute csrRowPtrC and nnzC */
	status = hipsparseSpruneDense2csrNnz(
		handle,
		m,
		n,
		d_A,
		lda,
		&threshold,
		descrC,
		d_csrRowPtrC,
		&nnzC, /* host */
		d_work);
	assert(HIPSPARSE_STATUS_SUCCESS == status);
	cudaStat1 = hipDeviceSynchronize();
	assert(hipSuccess == cudaStat1);
	printf("nnzC = %d\n", nnzC);
	if (0 == nnzC) {
		printf("C is empty \n");
		return 0;
	}
	/* step 5: compute csrColIndC and csrValC */
	cudaStat1 = hipMalloc((void**)&d_csrColIndC, sizeof(int) * nnzC);
	cudaStat2 = hipMalloc((void**)&d_csrValC, sizeof(float) * nnzC);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	status = hipsparseSpruneDense2csr(
		handle,
		m,
		n,
		d_A,
		lda,
		&threshold,
		descrC,
		d_csrValC,
		d_csrRowPtrC,
		d_csrColIndC,
		d_work);
	assert(HIPSPARSE_STATUS_SUCCESS == status);
	cudaStat1 = hipDeviceSynchronize();
	assert(hipSuccess == cudaStat1);
	/* step 6: output C */
	csrRowPtrC = (int*)malloc(sizeof(int) * (m + 1));
	csrColIndC = (int*)malloc(sizeof(int) * nnzC);
	csrValC = (float*)malloc(sizeof(float) * nnzC);
	assert(NULL != csrRowPtrC);
	assert(NULL != csrColIndC);
	assert(NULL != csrValC);
	cudaStat1 = hipMemcpy(csrRowPtrC, d_csrRowPtrC, sizeof(int) * (m + 1),
		hipMemcpyDeviceToHost);
	cudaStat2 = hipMemcpy(csrColIndC, d_csrColIndC, sizeof(int) * nnzC,
		hipMemcpyDeviceToHost);
	cudaStat3 = hipMemcpy(csrValC, d_csrValC, sizeof(float) * nnzC,
		hipMemcpyDeviceToHost);
	assert(hipSuccess == cudaStat1);
	assert(hipSuccess == cudaStat2);
	assert(hipSuccess == cudaStat3);
	printCsr(m, n, nnzC, descrC, csrValC, csrRowPtrC, csrColIndC, "C");
	/* free resources */
	if (d_A) hipFree(d_A);
	if (d_csrRowPtrC) hipFree(d_csrRowPtrC);
	if (d_csrColIndC) hipFree(d_csrColIndC);
	if (d_csrValC) hipFree(d_csrValC);
	if (csrRowPtrC) free(csrRowPtrC);
	if (csrColIndC) free(csrColIndC);
	if (csrValC) free(csrValC);
	if (handle) hipsparseDestroy(handle);
	if (stream) hipStreamDestroy(stream);
	if (descrC) hipsparseDestroyMatDescr(descrC);
	hipDeviceReset();
	return 0;
}