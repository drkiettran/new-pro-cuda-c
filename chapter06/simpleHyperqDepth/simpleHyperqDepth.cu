#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

/*
 * This example demonstrates submitting work to a CUDA stream in depth-first
 * order. Work submission in depth-first order may introduce false-dependencies
 * between unrelated tasks in different CUDA streams, limiting the parallelism
 * of a CUDA application. kernel_1, kernel_2, kernel_3, and kernel_4 simply
 * implement identical, dummy computation. Separate kernels are used to make the
 * scheduling of these kernels simpler to visualize in the Visual Profiler.
 */

#define N 300000
#define NSTREAM 4

__global__ void kernel_1()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_2()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_3()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

__global__ void kernel_4()
{
    double sum = 0.0;

    for (int i = 0; i < N; i++)
    {
        sum = sum + tan(0.1) * tan(0.1);
    }
}

int main(int argc, char** argv)
{
    setbuf(stdout, NULL); // disable buffering.
    printf("Run program as follows:\n");
    printf("%s [num streams] [bigcase (0/1)]\n\n", argv[0]);

    int n_streams = NSTREAM;
    int isize = 1;
    int iblock = 1;
    int bigcase = 0;

    // get argument from command line
    if (argc > 1) n_streams = atoi(argv[1]);

    if (argc > 2) bigcase = atoi(argv[2]);

    std::cout << "n_streams: " << n_streams << std::endl;
    std::cout << "bigcase: " << bigcase << std::endl;
    float elapsed_time;

    // set up max connection
    char* iname = "CUDA_DEVICE_MAX_CONNECTIONS";
    // setenv(iname, "32", 1); UNIX ONLY. In the Debugging settings, set the environment var there
    //_putenv(strcat(iname,"=32"));

    char* ivalue = getenv(iname);
    printf("%s = %s\n", iname, ivalue);

    std::cout << iname << "=" << ivalue << std::endl;
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("> Using Device %d: %s with num_streams=%d\n", dev, deviceProp.name, n_streams);
    CHECK(hipSetDevice(dev));

    // check if device support hyper-q
    if (deviceProp.major < 3 || (deviceProp.major == 3 && deviceProp.minor < 5))
    {
        if (deviceProp.concurrentKernels == 0)
        {
            printf("> GPU does not support concurrent kernel execution (SM 3.5 "
                "or higher required)\n");
            printf("> CUDA kernel runs will be serialized\n");
        }
        else
        {
            printf("> GPU does not support HyperQ\n");
            printf("> CUDA kernel runs will have limited concurrency\n");
        }
    }

    printf("> Compute Capability %d.%d hardware with %d multi-processors\n",
        deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

    // Allocate and initialize an array of stream handles
    hipStream_t* streams = (hipStream_t*)malloc(n_streams * sizeof(
        hipStream_t));

    for (int i = 0; i < n_streams; i++)
    {
        CHECK(hipStreamCreate(&(streams[i])));
    }

    // run kernel with more threads
    if (bigcase == 1)
    {
        iblock = 512;
        isize = 1 << 12;
    }

    // set up execution configuration
    dim3 block(iblock);
    dim3 grid(isize / iblock);
    printf("> grid %d block %d\n", grid.x, block.x);

    // creat events
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // record start event
    CHECK(hipEventRecord(start, 0));

    // dispatch job with depth first ordering
    for (int i = 0; i < n_streams; i++)
    {
        kernel_1 << <grid, block, 0, streams[i] >> > ();
        kernel_2 << <grid, block, 0, streams[i] >> > ();
        kernel_3 << <grid, block, 0, streams[i] >> > ();
        kernel_4 << <grid, block, 0, streams[i] >> > ();
    }

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution = %.3fs\n",
        elapsed_time / 1000.0f);

    // release all stream
    for (int i = 0; i < n_streams; i++)
    {
        CHECK(hipStreamDestroy(streams[i]));
    }

    free(streams);

    // destroy events
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));

    // reset device
    CHECK(hipDeviceReset());

    return 0;
}
