#include "hip/hip_runtime.h"
/* File:     vec_add.cu
 * Purpose:  Implement vector addition on a gpu using cuda
 *
 * Compile:  nvcc [-g] [-G] -arch=sm_21 -o vec_add vec_add.cu 
 * Run:      ./vec_add <n>
 *              n is the vector length
 *
 * Input:    None
 * Output:   Result of vector addition.  If all goes well it should
 *           be a vector consisting of n copies of n+1.
 *
 * Notes:
 * 1.  CUDA is installed on all of the machines in HR 530, HR 235, and
 *     and LS G12
 * 2.  If you get something like "nvcc: command not found" when you try
 *     to compile your program.  Type the following command
 *
 *           $ export PATH=/usr/local/cuda/bin:$PATH
 *
 *     (As usual the "$" is the shell prompt:  just type the rest 
 *     of the line.)
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/* Kernel for vector addition */
__global__ void Vec_add(float x[], float y[], float z[], int n) {
   /* blockDim.x = threads_per_block                            */
   /* First block gets first threads_per_block components.      */
   /* Second block gets next threads_per_block components, etc. */
   int i = blockDim.x * blockIdx.x + threadIdx.x;

   /* block_count*threads_per_block may be >= n */
   if (i < n) z[i] = x[i] + y[i];
}  /* Vec_add */


/* Host code */
int main(int argc, char* argv[]) {
   int n, i;
   float *h_x, *h_y, *h_z;
   float *d_x, *d_y, *d_z;
   int threads_per_block;
   int block_count;
   size_t size;

   /* Get number of components in vector */
   if (argc != 2) {
      fprintf(stderr, "usage: %s <vector order>\n", argv[0]);
      exit(0);
   }
   n = strtol(argv[1], NULL, 10);
   size = n*sizeof(float);

   /* Allocate input vectors in host memory */
   h_x = (float*) malloc(size);
   h_y = (float*) malloc(size);
   h_z = (float*) malloc(size);
   
   /* Initialize input vectors */
   for (i = 0; i < n; i++) {
      h_x[i] = i+1;
      h_y[i] = n-i;
   }

   printf("h_x = ");
   for (i = 0; i < n; i++)
      printf("%.1f ", h_x[i]);
   printf("\n");

   printf("h_y = ");
   for (i = 0; i < n; i++)
      printf("%.1f ", h_y[i]);
   printf("\n");

   /* Allocate vectors in device memory */
   hipMalloc(&d_x, size);
   hipMalloc(&d_y, size);
   hipMalloc(&d_z, size);

   /* Copy vectors from host memory to device memory */
   hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
   hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

   /* Define block size */
   threads_per_block = 256;

   /* Define grid size.  If we just computed n/threads_per_block */
   /* we might get fewer threads than vector components.  Using  */
   /* ceil(n/threads_per_block) guarantees at least one thread   */
   /* per vector component.  The following formula is a kludge   */
   /* since it appears that the CUDA ceil function doesn't work  */
   /* correctly.                                                 */
   block_count = (n + threads_per_block - 1)/threads_per_block;

   /* Invoke kernel using block_count blocks, each of which  */
   /* contains threads_per_block threads                     */
   Vec_add<<<block_count, threads_per_block>>>(d_x, d_y, d_z, n);

   /* Wait for the kernel to complete */
   hipDeviceSynchronize();

   /* Copy result from device memory to host memory */
   /* h_z contains the result in host memory        */
   hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);

   printf("The sum is: \n");
   for (i = 0; i < n; i++)
      printf("%.1f ", h_z[i]);
   printf("\n");

   /* Free device memory */
   hipFree(d_x);
   hipFree(d_y);
   hipFree(d_z);

   /* Free host memory */
   free(h_x);
   free(h_y);
   free(h_z);

   return 0;
}  /* main */