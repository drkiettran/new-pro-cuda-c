#include "hip/hip_runtime.h"
﻿#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include <tclap/CmdLine.h>

/*
 * This example demonstrates a simple vector sum on the GPU and on the host.
 * sumArraysOnGPU splits the work of the vector sum across CUDA threads on the
 * GPU. Only a single thread block is used in this small case, for simplicity.
 * sumArraysOnHost sequentially iterates through vector elements on the host.
 * This version of sumArrays adds host timers to measure GPU and CPU
 * performance.
 */

void checkResult(float* hostRef, float* gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i],
                gpuRef[i], i);
            break;
        }
    }

    if (match) printf("Arrays match.\n\n");

    return;
}

void initialData(float* ip, int size)
{
    // generate different seed for random number
    // time_t t;
    // srand((unsigned)time(&t));

    for (int i = 0; i < size; i++)
    {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }

    return;
}

void sumArraysOnHost(float* A, float* B, float* C, const int N)
{
    for (int idx = 0; idx < N; idx++)
    {
        C[idx] = A[idx] + B[idx];
    }
}
__global__ void sumArraysOnGPU(float* A, float* B, float* C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) C[i] = A[i] + B[i];
}

void getArgs(int argc, char** argv, int& n, int& b) {
    try {
        TCLAP::CmdLine cmd("MyProgram - A sample C++ program", ' ', "1.0");

        TCLAP::ValueArg<int> nArg("n", "num-elem", "Number of elements", false, 512, "int");
        TCLAP::ValueArg<int> bArg("b", "block-size", "Number of threads per block", false, 512, "int");
        cmd.add(bArg);
        cmd.add(nArg);
        cmd.parse(argc, argv);
        b = bArg.getValue();
        n = nArg.getValue();
    }
    catch (TCLAP::ArgException& e) {
        std::cerr << "Error: " << e.error() << " for argument " << e.argId() << std::endl;
    }
}

int main(int argc, char** argv)
{
    int block_x, n;
    std::chrono::steady_clock::time_point begin;
    getArgs(argc, argv, n, block_x);
    printf("%s Starting... with block.x = %d\n", argv[0], block_x);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of vectors
    printf("Vector size %d\n", n);

    // malloc host memory
    size_t nBytes = n * sizeof(float);

    float* h_A, * h_B, * hostRef, * gpuRef;
    h_A = (float*)malloc(nBytes);
    h_B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);

    // initialize data at host side
    begin = StartTimer();
    initialData(h_A, n);
    initialData(h_B, n);
    std::cout << "Initialize Arrays on Host: " << GetDurationInMilliSeconds(begin, StopTimer()) << " ms" << std::endl;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add vector at host side for result checks
    begin = StartTimer();
    sumArraysOnHost(h_A, h_B, hostRef, n);
    std::cout << "Sum Arrays on Host: " << GetDurationInMilliSeconds(begin, StopTimer()) << " ms" << std::endl;

    // malloc device global memory
    float* d_A, * d_B, * d_C;
    CHECK(hipMalloc((float**)&d_A, nBytes));
    CHECK(hipMalloc((float**)&d_B, nBytes));
    CHECK(hipMalloc((float**)&d_C, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_C, gpuRef, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int iLen = block_x;
    dim3 block(iLen);
    dim3 grid((n + block.x - 1) / block.x);

    begin = StartTimer();
    sumArraysOnGPU << <grid, block >> > (d_A, d_B, d_C, n);
    std::cout << "Sum Arrays on GPU: " << GetDurationInMilliSeconds(begin, StopTimer()) << " ms" << std::endl;

    CHECK(hipDeviceSynchronize());
    // check kernel error
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, n);

    // free device global memory
    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return(0);
}
