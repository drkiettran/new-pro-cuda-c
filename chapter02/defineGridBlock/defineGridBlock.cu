﻿
#include <stdlib.h>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"

/*
 * Demonstrate defining the dimensions of a block of threads and a grid of
 * blocks from the host.
 */

int main(int argc, char** argv)
{
    // define total data element
    int nElem = 1024;
    if (argc > 1) {
        nElem = atoi(argv[1]);
    }

    // define grid and block structure
    dim3 block(1024);
    dim3 grid((nElem + block.x - 1) / block.x);
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset block
    block.x = 512;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset block
    block.x = 256;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset block
    block.x = 128;
    grid.x = (nElem + block.x - 1) / block.x;
    printf("grid.x %d block.x %d \n", grid.x, block.x);

    // reset device before you leave
    CHECK(hipDeviceReset());

    return(0);
}


