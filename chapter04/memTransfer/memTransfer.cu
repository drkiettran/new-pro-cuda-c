﻿#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "Counter.h"

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}
/*
 * An example of using CUDA's memory copy API to transfer data to and from the
 * device. In this case, hipMalloc is used to allocate memory on the GPU and
 * hipMemcpy is used to transfer the contents of host memory to an array
 * allocated using hipMalloc.
 */

int main(int argc, char** argv)
{
    // set up device
    int dev = 0;
    CHECK(hipSetDevice(dev));

    // memory size
    unsigned int isize = 1 << 8; // 22;
    unsigned int nbytes = isize * sizeof(float);

    // get device information
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting at ", argv[0]);
    printf("device %d: %s memory size %d nbyte %5.2fMB\n", dev,
        deviceProp.name, isize, nbytes / (1024.0f * 1024.0f));

    // allocate the host memory
    float* h_a = (float*)malloc(nbytes);

    // allocate the device memory
    float* d_a;
    CHECK(hipMalloc((float**)&d_a, nbytes));

    // initialize the host memory
    for (unsigned int i = 0; i < isize; i++) h_a[i] = 0.5f;

    // transfer data from the host to the device
    CHECK(hipMemcpy(d_a, h_a, nbytes, hipMemcpyHostToDevice));

    // transfer data from the device to the host
    CHECK(hipMemcpy(h_a, d_a, nbytes, hipMemcpyDeviceToHost));

    // free memory
    CHECK(hipFree(d_a));
    free(h_a);

    // reset device
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}
